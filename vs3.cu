/*
 * EEDI2CUDA: EEDI2 filter using CUDA
 *
 * Copyright (C) 2005-2006 Kevin Stone
 * Copyright (C) 2014-2019 HolyWu
 * Copyright (C) 2021 Misaki Kasumi
 *
 * This program is free software; you can redistribute it and/or modify it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along with this program; if not, write to the Free Software Foundation,
 * Inc., 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA
 */

#include <algorithm>
#include <atomic>
#include <memory>
#include <optional>
#include <stdexcept>
#include <tuple>
#include <utility>
#include <vector>

#include <boost/sync/semaphore.hpp>

#include <VSHelper.h>
#include <VapourSynth.h>

#include "config.h"

#include "eedi2.cuh"
#include "utils.cuh"

using namespace std::literals::string_literals;

struct PropsMap : public std::multimap<std::string_view, int64_t> {
  using std::multimap<std::string_view, int64_t>::multimap;

  std::optional<mapped_type> get(const key_type &key, const size_type idx = static_cast<size_type>(-1)) const {
    auto casual = idx == static_cast<size_type>(-1);
    auto [bg, ed] = casual ? std::make_pair(find(key), end()) : equal_range(key);
    if (bg == ed)
      return std::nullopt;
    if (!casual)
      for (size_type i = 0; i < idx; ++i)
        if (++bg == ed)
          return std::nullopt;
    return std::make_optional(bg->second);
  }
};

template <typename T> class BasePipeline {
protected:
  std::vector<std::unique_ptr<Pass<T>>> passes;
  VideoInfo vi;
  int device_id;
  hipStream_t stream;
  T *h_src, *h_dst;
  std::vector<T *> fbs;

protected:
  VideoInfo getOutputVI() const { return passes.back()->getOutputVI(); }

public:
  BasePipeline(std::string_view filterName, const PropsMap &props, VideoInfo vi) : vi(vi) {
    using invalid_arg = std::invalid_argument;

    auto vi2 = vi;
    EEDI2Param d;
    unsigned map, pp, fieldS;

    if (vi.width < 8 || vi.height < 7)
      throw invalid_arg("clip resolution too low");

    if (filterName == "EEDI2")
      numeric_cast_to(fieldS, props.get("field").value());
    else
      fieldS = 1;

    numeric_cast_to(d.mthresh, props.get("mthresh").value_or(10));
    numeric_cast_to(d.lthresh, props.get("lthresh").value_or(20));
    numeric_cast_to(d.vthresh, props.get("vthresh").value_or(20));

    numeric_cast_to(d.estr, props.get("estr").value_or(2));
    numeric_cast_to(d.dstr, props.get("dstr").value_or(4));
    numeric_cast_to(d.maxd, props.get("maxd").value_or(24));

    numeric_cast_to(map, props.get("map").value_or(0));
    numeric_cast_to(pp, props.get("pp").value_or(1));

    unsigned nt;
    numeric_cast_to(nt, props.get("nt").value_or(50));

    numeric_cast_to(device_id, props.get("device_id").value_or(-1));

    if (fieldS > 3)
      throw invalid_arg("field must be 0, 1, 2 or 3");
    if (d.maxd < 1 || d.maxd > 29)
      throw invalid_arg("maxd must be between 1 and 29 (inclusive)");
    if (map > 3)
      throw invalid_arg("map must be 0, 1, 2 or 3");
    if (pp > 1)
      throw invalid_arg("only pp=0 or 1 is implemented");

    if (map == 0 || map == 3)
      vi2.height *= 2;

    d.mthresh *= d.mthresh;
    d.vthresh *= 81;

    nt <<= sizeof(T) * 8 - 8;
    d.nt4 = nt * 4;
    d.nt7 = nt * 7;
    d.nt8 = nt * 8;
    d.nt13 = nt * 13;
    d.nt19 = nt * 19;

    passes.emplace_back(new EEDI2Pass<T>(vi, vi2, d, map, pp, fieldS));

    if (filterName != "EEDI2") {
      auto vi3 = vi2;
      std::swap(vi3.width, vi3.height); // XXX: this is correct for 420 & 444 only
      passes.emplace_back(new TransposePass<T>(vi2, vi3));
      auto vi4 = vi3;
      if (filterName == "AA2") {
        vi4.width /= 2;
        passes.emplace_back(new ScaleDownWPass<T>(vi3, vi4));
      } else {
        passes.emplace_back(new ShiftWPass<T>(vi3, vi4));
      }
      auto vi5 = vi4;
      vi5.height *= 2;
      passes.emplace_back(new EEDI2Pass<T>(vi4, vi5, d, map, pp, fieldS));
      auto vi6 = vi5;
      std::swap(vi6.width, vi6.height);
      passes.emplace_back(new TransposePass<T>(vi5, vi6));
      auto vi7 = vi6;
      if (filterName == "AA2") {
        vi7.width /= 2;
        passes.emplace_back(new ScaleDownWPass<T>(vi6, vi7));
      } else {
        passes.emplace_back(new ShiftWPass<T>(vi6, vi7));
      }
    }

    passes.shrink_to_fit();

    initCuda();
  }

  BasePipeline(const BasePipeline &other) : vi(other.vi), device_id(other.device_id) {
    passes.reserve(other.passes.size());
    for (const auto &step : other.passes)
      passes.emplace_back(step->dup());

    initCuda();
  }

  ~BasePipeline() {
    try_cuda(hipHostFree(h_src));
    try_cuda(hipHostFree(h_dst));
    for (auto fb : fbs)
      try_cuda(hipFree(fb));
  }

private:
  void initCuda() {
    try {
      try_cuda(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    } catch (const CUDAError &exc) {
      throw CUDAError(exc.what() + " Please upgrade your driver."s);
    }

    if (auto &firstStep = *passes.front(); !firstStep.getSrcDevPtr()) {
      size_t pitch;
      T *fb_d_src;
      try_cuda(hipMallocPitch(&fb_d_src, &pitch, vi.width * sizeof(T), vi.height));
      firstStep.setSrcDevPtr(fb_d_src);
      firstStep.setSrcPitch(static_cast<unsigned>(pitch));
      fbs.push_back(fb_d_src);
    }

    if (auto &lastStep = *passes.back(); !lastStep.getDstDevPtr()) {
      auto vi2 = lastStep.getOutputVI();
      size_t pitch;
      T *fb_d_dst;
      try_cuda(hipMallocPitch(&fb_d_dst, &pitch, vi2.width * sizeof(T), vi2.height));
      lastStep.setDstDevPtr(fb_d_dst);
      lastStep.setDstPitch(static_cast<unsigned>(pitch));
      fbs.push_back(fb_d_dst);
    }

    auto d_pitch_src = passes.front()->getSrcPitch();
    auto d_pitch_dst = passes.back()->getDstPitch();
    auto src_height = vi.height;
    auto dst_height = passes.back()->getOutputVI().height;
    try_cuda(hipHostAlloc(&h_src, d_pitch_src * src_height, hipHostMallocWriteCombined));
    try_cuda(hipHostAlloc(&h_dst, d_pitch_dst * dst_height, hipHostMallocDefault));
  }
};

VideoInfo get_vi(const VSMap *in, const VSAPI *vsapi) {
  auto node = vsapi->propGetNode(in, "clip", 0, nullptr);
  auto vi = vsapi->getVideoInfo(node);
  vsapi->freeNode(node);
  VideoInfo vvi{vi->width, vi->height, vi->format->subSamplingW};
  return vvi;
}

PropsMap mapize(const VSMap *in, const VSAPI *vsapi) {
  PropsMap m;
  for (auto i = 0, num_keys = vsapi->propNumKeys(in); i < num_keys; ++i) {
    auto key = vsapi->propGetKey(in, i);
    if (vsapi->propGetType(in, key) != ptInt)
      continue;
    auto num_el = vsapi->propNumElements(in, key);
    for (auto j = 0; j < num_el; ++j) {
      auto val = vsapi->propGetInt(in, key, j, nullptr);
      m.emplace(key, val);
    }
  }
  return m;
}

template <typename T> class Pipeline : public BasePipeline<T> {
  std::unique_ptr<VSNodeRef, void(VS_CC *const)(VSNodeRef *)> node;
  VSVideoInfo vi2;

public:
  Pipeline(std::string_view filterName, const VSMap *in, const VSAPI *vsapi)
      : BasePipeline<T>(filterName, mapize(in, vsapi), get_vi(in, vsapi)),
        node(vsapi->propGetNode(in, "clip", 0, nullptr), vsapi->freeNode) {
    vi2 = *vsapi->getVideoInfo(node.get());
    auto ovi = BasePipeline<T>::getOutputVI();
    vi2.width = ovi.width;
    vi2.height = ovi.height;
  }

  Pipeline(const Pipeline &other, const VSAPI *vsapi)
      : BasePipeline<T>(other), node(vsapi->cloneNodeRef(other.node.get()), vsapi->freeNode), vi2(other.vi2) {}

  const VSVideoInfo &getOutputVI() const { return vi2; }

  VSFrameRef *getFrame(int n, int activationReason, VSFrameContext *frameCtx, VSCore *core, const VSAPI *vsapi) {
    if (activationReason == arInitial) {
      vsapi->requestFrameFilter(n, node.get(), frameCtx);
      return nullptr;
    } else if (activationReason != arAllFramesReady)
      return nullptr;

    if (device_id != -1)
      try_cuda(hipSetDevice(device_id));

    std::unique_ptr<const VSFrameRef, void(VS_CC *const)(const VSFrameRef *)> src_frame{vsapi->getFrameFilter(n, node.get(), frameCtx),
                                                                                        vsapi->freeFrame};
    std::unique_ptr<VSFrameRef, void(VS_CC *const)(const VSFrameRef *)> dst_frame{
        vsapi->newVideoFrame(vi2.format, vi2.width, vi2.height, src_frame.get(), core), vsapi->freeFrame};

    for (int plane = 0; plane < vi2.format->numPlanes; ++plane) {
      auto src_width = vsapi->getFrameWidth(src_frame.get(), plane);
      auto src_height = vsapi->getFrameHeight(src_frame.get(), plane);
      auto dst_width = vsapi->getFrameWidth(dst_frame.get(), plane);
      auto dst_height = vsapi->getFrameHeight(dst_frame.get(), plane);
      auto s_pitch_src = vsapi->getStride(src_frame.get(), plane);
      auto s_pitch_dst = vsapi->getStride(dst_frame.get(), plane);
      auto src_width_bytes = src_width * sizeof(T);
      auto dst_width_bytes = dst_width * sizeof(T);
      auto s_src = vsapi->getReadPtr(src_frame.get(), plane);
      auto s_dst = vsapi->getWritePtr(dst_frame.get(), plane);
      auto d_src = passes.front()->getSrcDevPtr();
      auto d_dst = passes.back()->getDstDevPtr();
      auto d_pitch_src = passes.front()->getSrcPitch() >> !!plane * vi2.format->subSamplingW;
      auto d_pitch_dst = passes.back()->getDstPitch() >> !!plane * vi2.format->subSamplingW;

      // upload
      vs_bitblt(h_src, d_pitch_src, s_src, s_pitch_src, src_width_bytes, src_height);
      try_cuda(hipMemcpy2DAsync(d_src, d_pitch_src, h_src, d_pitch_src, src_width_bytes, src_height, hipMemcpyHostToDevice, stream));

      // process
      for (unsigned i = 0; i < passes.size(); ++i) {
        auto &cur = *passes[i];
        if (i) {
          auto &last = *passes[i - 1];
          auto &next = *passes[i + 1];
          auto last_vi = last.getOutputVI();
          auto ss = !!plane * last_vi.subSampling;
          if (!cur.getSrcDevPtr()) {
            cur.setSrcDevPtr(const_cast<T *>(last.getDstDevPtr()));
            cur.setSrcPitch(last.getDstPitch());
          }
          if (!cur.getDstDevPtr()) {
            cur.setDstDevPtr(next.getSrcDevPtr());
            cur.setDstPitch(next.getSrcPitch());
          }
          if (!cur.getDstDevPtr()) {
            auto vi = cur.getOutputVI();
            size_t pitch;
            T *fb;
            try_cuda(hipMallocPitch(&fb, &pitch, vi.width * sizeof(T), vi.height));
            cur.setDstDevPtr(fb);
            next.setSrcDevPtr(fb);
            cur.setDstPitch(static_cast<unsigned>(pitch));
            next.setSrcPitch(static_cast<unsigned>(pitch));
            fbs.push_back(fb);
          }
          auto curPtr = cur.getSrcDevPtr();
          auto lastPtr = last.getDstDevPtr();
          if (curPtr != lastPtr)
            try_cuda(hipMemcpy2DAsync(curPtr, cur.getSrcPitch() >> ss, lastPtr, last.getDstPitch() >> ss, last_vi.width * sizeof(T) >> ss,
                                       last_vi.height >> ss, hipMemcpyDeviceToDevice, stream));
        }
        cur.process(n, plane, stream);
      }

      // download
      try_cuda(hipMemcpy2DAsync(h_dst, d_pitch_dst, d_dst, d_pitch_dst, dst_width_bytes, dst_height, hipMemcpyDeviceToHost, stream));
      try_cuda(hipStreamSynchronize(stream));
      vs_bitblt(s_dst, s_pitch_dst, h_dst, d_pitch_dst, dst_width_bytes, dst_height);
    }

    return dst_frame.release();
  }
};

template <typename T> class Instance {
  using Item = std::pair<Pipeline<T>, std::atomic_flag>;
  boost::sync::semaphore semaphore;

  inline Item *items() noexcept { return reinterpret_cast<Item *>(reinterpret_cast<unsigned *>(this + 1) + 1); }
  inline unsigned num_streams() const noexcept { return *reinterpret_cast<const unsigned *>(this + 1); }

public:
  Instance(std::string_view filterName, const VSMap *in, const VSAPI *vsapi) : semaphore(num_streams()) {
    auto items = this->items();
    new (items) Item(std::piecewise_construct, std::forward_as_tuple(filterName, in, vsapi), std::forward_as_tuple());
    items[0].second.clear();
    for (unsigned i = 1; i < num_streams(); ++i) {
      new (items + i) Item(std::piecewise_construct, std::forward_as_tuple(firstReactor(), vsapi), std::forward_as_tuple());
      items[i].second.clear();
    }
  }

  ~Instance() {
    auto items = this->items();
    for (unsigned i = 0; i < num_streams(); ++i)
      items[i].~Item();
  }

  Pipeline<T> &firstReactor() { return items()[0].first; }

  Pipeline<T> &acquireReactor() {
    if (num_streams() == 1)
      return firstReactor();
    semaphore.wait();
    auto items = this->items();
    for (unsigned i = 0; i < num_streams(); ++i) {
      if (!items[i].second.test_and_set())
        return items[i].first;
    }
    unreachable();
  }

  void releaseReactor(const Pipeline<T> &instance) {
    if (num_streams() == 1)
      return;
    auto items = this->items();
    for (unsigned i = 0; i < num_streams(); ++i) {
      if (&instance == &items[i].first) {
        items[i].second.clear();
        break;
      }
    }
    semaphore.post();
  }

  static void *operator new(size_t sz, unsigned num_streams) {
    auto p = static_cast<Instance *>(::operator new(sz + sizeof(unsigned) + sizeof(Item) * num_streams));
    *reinterpret_cast<unsigned *>(p + 1) = num_streams;
    return p;
  }

  static void operator delete(void *p, unsigned) { ::operator delete(p); }

  static void operator delete(void *p) { ::operator delete(p); }
};

template <typename T> void VS_CC eedi2Init(VSMap *, VSMap *, void **instanceData, VSNode *node, VSCore *, const VSAPI *vsapi) {
  auto data = static_cast<Instance<T> *>(*instanceData);
  vsapi->setVideoInfo(&data->firstReactor().getOutputVI(), 1, node);
}

template <typename T>
const VSFrameRef *VS_CC eedi2GetFrame(int n, int activationReason, void **instanceData, void **, VSFrameContext *frameCtx, VSCore *core,
                                      const VSAPI *vsapi) {

  auto data = static_cast<Instance<T> *>(*instanceData);
  const VSFrameRef *out = nullptr;

  if (activationReason == arInitial) {
    out = data->firstReactor().getFrame(n, activationReason, frameCtx, core, vsapi);
  } else {
    auto &d = data->acquireReactor();
    try {
      out = d.getFrame(n, activationReason, frameCtx, core, vsapi);
    } catch (const std::exception &exc) {
      vsapi->setFilterError(("EEDI2CUDA: "s + exc.what()).c_str(), frameCtx);
    }
    data->releaseReactor(d);
  }

  return out;
}

template <typename T> void VS_CC eedi2Free(void *instanceData, VSCore *, const VSAPI *) {
  auto data = static_cast<Instance<T> *>(instanceData);
  delete data;
}

template <typename T> void eedi2CreateInner(std::string_view filterName, const VSMap *in, VSMap *out, const VSAPI *vsapi, VSCore *core) {
  try {
    int err;
    unsigned num_streams;
    numeric_cast_to(num_streams, vsapi->propGetInt(in, "num_streams", 0, &err));
    if (err)
      num_streams = 1;
    auto data = new (num_streams) Instance<T>(filterName, in, vsapi);
    vsapi->createFilter(in, out, filterName.data(), eedi2Init<T>, eedi2GetFrame<T>, eedi2Free<T>,
                        num_streams > 1 ? fmParallel : fmParallelRequests, 0, data, core);
  } catch (const std::exception &exc) {
    vsapi->setError(out, ("EEDI2CUDA: "s + exc.what()).c_str());
    return;
  }
}

VS_CC void eedi2Create(const VSMap *in, VSMap *out, void *userData, VSCore *core, const VSAPI *vsapi) {
  std::string_view filterName{static_cast<const char *>(userData)};
  VSNodeRef *node = vsapi->propGetNode(in, "clip", 0, nullptr);
  const VSVideoInfo *vi = vsapi->getVideoInfo(node);
  vsapi->freeNode(node);
  if (vi->format->bytesPerSample == 1)
    eedi2CreateInner<uint8_t>(filterName, in, out, vsapi, core);
  else
    eedi2CreateInner<uint16_t>(filterName, in, out, vsapi, core);
}

void VS_CC BuildConfigCreate(const VSMap *, VSMap *out, void *, VSCore *, const VSAPI *vsapi) {
  vsapi->propSetData(out, "version", VERSION, -1, paAppend);
  vsapi->propSetData(out, "options", BUILD_OPTIONS, -1, paAppend);
  vsapi->propSetData(out, "timestamp", CONFIGURE_TIME, -1, paAppend);
  vsapi->propSetInt(out, "vsapi_version", VAPOURSYNTH_API_VERSION, paAppend);
}

#define eedi2_common_params                                                                                                                \
  "mthresh:int:opt;"                                                                                                                       \
  "lthresh:int:opt;"                                                                                                                       \
  "vthresh:int:opt;"                                                                                                                       \
  "estr:int:opt;"                                                                                                                          \
  "dstr:int:opt;"                                                                                                                          \
  "maxd:int:opt;"                                                                                                                          \
  "map:int:opt;"                                                                                                                           \
  "nt:int:opt;"                                                                                                                            \
  "pp:int:opt;"                                                                                                                            \
  "num_streams:int:opt;"                                                                                                                   \
  "device_id:int:opt"

VS_EXTERNAL_API(void)
VapourSynthPluginInit(VSConfigPlugin configFunc, VSRegisterFunction registerFunc, VSPlugin *plugin) {
  auto to_voidp = [](auto *p) { return const_cast<void *>(static_cast<const void *>(p)); };

  configFunc("club.amusement.eedi2cuda", "eedi2cuda", "EEDI2 filter using CUDA", VAPOURSYNTH_API_VERSION, 1, plugin);
  registerFunc("EEDI2",
               "clip:clip;"
               "field:int;" eedi2_common_params,
               eedi2Create, to_voidp("EEDI2"), plugin);
  registerFunc("Enlarge2", "clip:clip;" eedi2_common_params, eedi2Create, to_voidp("Enlarge2"), plugin);
  registerFunc("AA2", "clip:clip;" eedi2_common_params, eedi2Create, to_voidp("AA2"), plugin);
  registerFunc("BuildConfig", "", BuildConfigCreate, nullptr, plugin);
}
